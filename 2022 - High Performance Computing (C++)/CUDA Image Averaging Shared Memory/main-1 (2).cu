#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "omp.h"
#include "utils/pngio.h"

#define BLOCK_SIZE (16u)
#define FILTER_SIZE (5u)
#define TILE_SIZE (12U) // BLOCK_SIZE - (2 * (FILTER_SIZE/2))



//defining kernel function
__global__ void processImage(unsigned char * out, const unsigned char * in, size_t pitch, unsigned int width, unsigned int height){
	
	int x_o = (TILE_SIZE*blockIdx.x) + threadIdx.x;
	int y_o = (TILE_SIZE*blockIdx.y) + threadIdx.y;
	
	int x_i = x_o - (FILTER_SIZE/2);
	int y_i = y_o - (FILTER_SIZE/2);
	int sum = 0;
	
	//defining shared memory
	__shared__ unsigned char sBuffer[BLOCK_SIZE][BLOCK_SIZE];
	
	//copying inside shared memory
	if ((x_i >= 0) && (x_i < width) && (y_i >= 0) && (y_i < width)){
		sBuffer[threadIdx.y][threadIdx.x] = in[y_i * pitch + x_i];
	}
	else
		sBuffer[threadIdx.y][threadIdx.x] = 0;
	
	__syncthreads();
	
	if ( threadIdx.x < TILE_SIZE && threadIdx.y < TILE_SIZE ) {
		for (int r = 0; r < FILTER_SIZE; ++r )
			for (int c = 0; c < FILTER_SIZE; ++c)
				sum += sBuffer[threadIdx.y + r][threadIdx.x + c];
	sum /= FILTER_SIZE * FILTER_SIZE;
	
	if ( x_o < width && y_o < height )
		out[ y_o * width + x_o] = sum;
				
		}
	}

int main(int argc, char **argv) {
	std::cout << "Loading image..." << std::endl;
	//loading function
	png::image<png::rgb_pixel> img("../lena.png");
	
	unsigned int width = img.get_width();
	unsigned int height = img.get_height();
	
	//defining size to allocate memory
	int size = width * height * sizeof(unsigned char);
	
	//allocating memory buffer to host memory
	unsigned char *h_r = (unsigned char*) malloc ( size * sizeof(unsigned char));
	unsigned char *h_g = (unsigned char*) malloc ( size * sizeof(unsigned char));
	unsigned char *h_b = (unsigned char*) malloc ( size * sizeof(unsigned char));
	
	//allocating memory for the output
	unsigned char *h_r_n = (unsigned char*) malloc ( size * sizeof(unsigned char));
	unsigned char *h_g_n = (unsigned char*) malloc ( size * sizeof(unsigned char));
	unsigned char *h_b_n = (unsigned char*) malloc ( size * sizeof(unsigned char));
	
	//converting image to raw buffer
	pvg::pngtoRgb3(h_r, h_g, h_b, img);
	
	//allocating memory on device
	unsigned char *d_r_n = NULL;
	unsigned char *d_g_n = NULL;
	unsigned char *d_b_n = NULL;
	
	CUDA_CHECK_RETURN(hipMalloc(&d_r_n, size));
	CUDA_CHECK_RETURN(hipMalloc(&d_g_n, size));
	CUDA_CHECK_RETURN(hipMalloc(&d_b_n, size));
	
	//allocating image buffer on device
	
	unsigned char *d_r = NULL;
	unsigned char *d_g = NULL;
	unsigned char *d_b = NULL;
	
	size_t pitch_r = 0;
	size_t pitch_g = 0;
	size_t pitch_b = 0;
	
	CUDA_CHECK_RETURN( hipMallocPitch(&d_r, &pitch_r, width, height));
	CUDA_CHECK_RETURN( hipMallocPitch(&d_g, &pitch_g, width, height));
	CUDA_CHECK_RETURN( hipMallocPitch(&d_b, &pitch_b, width, height));
	
	//copy raw buffer from host to device
	CUDA_CHECK_RETURN( hipMemcpy2D(d_r, pitch_r, h_r, width, width, height, hipMemcpyHostToDevice) );
	CUDA_CHECK_RETURN( hipMemcpy2D(d_g, pitch_g, h_g, width, width, height, hipMemcpyHostToDevice) );
	CUDA_CHECK_RETURN( hipMemcpy2D(d_b, pitch_b, h_b, width, width, height, hipMemcpyHostToDevice) );
	
	//configure image kernel
	dim3 grid_size((width + TILE_SIZE - 1)/TILE_SIZE, (height + TILE_SIZE -1)/TILE_SIZE);
	dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
	
	double start = omp_get_wtime();
	
	processImage<<<grid_size, block_size>>>(d_r_n, d_r, pitch_r, width, height);
	processImage<<<grid_size, block_size>>>(d_g_n, d_g, pitch_g, width, height);
	processImage<<<grid_size, block_size>>>(d_b_n, d_b, pitch_b, width, height);
	
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	
	double end = omp_get_wtime();
	
	CUDA_CHECK_RETURN( hipMemcpy(h_r_n, d_r_n, size, hipMemcpyDeviceToHost) );
	CUDA_CHECK_RETURN( hipMemcpy(h_g_n, d_g_n, size, hipMemcpyDeviceToHost) );
	CUDA_CHECK_RETURN( hipMemcpy(h_g_n, d_b_n, size, hipMemcpyDeviceToHost) );
	
	pvg::rgb3ToPng(img, h_r_n, h_g_n, h_b_n);
	std::cout<< "Done in "	<< end-start << " seconds" << std::endl;
	
	img.write("../lena_new.png");
	
	CUDA_CHECK_RETURN(hipFree(d_r));
	CUDA_CHECK_RETURN(hipFree(d_r_n));
	
	CUDA_CHECK_RETURN(hipFree(d_g));
	CUDA_CHECK_RETURN(hipFree(d_g_n));
	
	CUDA_CHECK_RETURN(hipFree(d_b));
	CUDA_CHECK_RETURN(hipFree(d_b_n));
	
	free(h_r);
	free(h_r_n);
	
	free(h_g);
	free(h_g_n);
	
	free(h_b);
	free(h_b_n);
	
	return 0;
}
