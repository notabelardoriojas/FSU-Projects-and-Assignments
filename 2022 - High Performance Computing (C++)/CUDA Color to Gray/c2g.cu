#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "Jpegfile.h"
#include <time.h>


__global__
void avg(BYTE * buffer)
{
    int i = blockIdx.x; 

    BYTE *pRed, *pGrn, *pBlu;
    pRed = buffer + 3 * i;
    pGrn = buffer + 3 * i + 1;
    pBlu = buffer + 3 * i + 2;

    // luminance
    int lum = (int)(.299 * (double)(*pRed) + .587 * (double)(*pGrn) + .114 * (double)(*pBlu));

    *pRed = (BYTE)lum;
    *pGrn = (BYTE)lum;
    *pBlu = (BYTE)lum;
}

int main()
{
    UINT img_y;
    UINT img_x;
    BYTE *buffer;
    clock_t t;
    //start timing
    t = clock();
    
    //read the file to buffer with RGB format
    buffer = JpegFile::JpegFileToRGB("sample.jpg", &img_x, &img_y);

    //the following code convert RGB to gray luminance.
    std::cout << "Dimensions: " << img_x << " " << img_y << std::endl;

    BYTE * d_buffer;
    hipMalloc((void **)&d_buffer, img_x * img_y * 3 * sizeof(BYTE));

    hipMemcpy(d_buffer, buffer, img_x * img_y * 3 * sizeof(BYTE), hipMemcpyHostToDevice);

    avg<<<img_x * img_y, 1>>>(d_buffer);

    hipMemcpy(buffer, d_buffer, img_x * img_y * 3 * sizeof(BYTE), hipMemcpyDeviceToHost);

    //write the gray luminance to another jpg file
    JpegFile::RGBToJpegFile("mono.jpg", buffer, img_x, img_y, true, 75);
    
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds
    printf("Took %f seconds to execute \n", time_taken);
    
    delete buffer;
    hipFree(d_buffer);

	return 1;
}
